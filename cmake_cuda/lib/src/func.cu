#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "func.h"

using namespace std;

__global__ void add(int *a, const int *b){
	int i = blockIdx.x;
	a[i] += b[i];
}

extern "C"
int Enter::Add_Gpu(){
	const int N = 10;
	int *a, *b, *temp, i;
	temp = new int [N];
	hipMalloc(&a, N*sizeof(int));
	hipMalloc(&b, N*sizeof(int));
	
	for (i=0;i<N;i++) temp[i] = i;
	hipMemcpy(a, temp, N * sizeof(int), hipMemcpyHostToDevice);
	for (i=0;i<N;i++) temp[i] = 2 * i;
	hipMemcpy(b, temp, N * sizeof(int), hipMemcpyHostToDevice);
	add<<<N, 1>>>(a, b);
	hipMemcpy(temp, a, N * sizeof(int), hipMemcpyDeviceToHost);
	for (i=0;i<N;i++){
		cout << temp[i] <<endl;
	}
	delete [] temp;
	hipFree(a);
	hipFree(b);
	
	return 0;
}